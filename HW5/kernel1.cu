#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#define BLOCK_SIZE 16

__device__ int mandel(float c_re, float c_im, int count)
{
    float z_re = c_re, z_im = c_im;
    int i;
    for (i = 0; i < count; ++i)
    {
        if (z_re * z_re + z_im * z_im > 4.f)
        break;

        float new_re = z_re * z_re - z_im * z_im;
        float new_im = 2.f * z_re * z_im;
        z_re = c_re + new_re;
        z_im = c_im + new_im;
    }

    return i;
}


__global__ void mandelKernel(float lowerX, float lowerY, float resX, float resY, float stepX, float stepY, int *output, int maxIterations) {
    // To avoid error caused by the floating number, use the following pseudo code
    //
    // float x = lowerX + thisX * stepX;
    // float y = lowerY + thisY * stepY;
    int thisX = blockIdx.x * blockDim.x + threadIdx.x;
    int thisY = blockIdx.y * blockDim.y + threadIdx.y;

    if(thisX >= resX || thisY >= resY) return;

    float x = lowerX + thisX * stepX;
    float y = lowerY + thisY * stepY;

    int index = thisY * resX + thisX;
    output[index] = mandel(x, y, maxIterations);


}

// Host front-end function that allocates the memory and launches the GPU kernel
void hostFE (float upperX, float upperY, float lowerX, float lowerY, int* img, int resX, int resY, int maxIterations)
{
    float stepX = (upperX - lowerX) / resX;
    float stepY = (upperY - lowerY) / resY;
    int N = resX * resY;
    int size = N * sizeof(int);

    int *data;
    data = (int*) malloc(size);
    int *d_data;
    hipMalloc(&d_data, size);

    dim3 blockSize(BLOCK_SIZE, BLOCK_SIZE);
    dim3 numBlocks(resX / blockSize.x, resY / blockSize.y);
    mandelKernel<<<numBlocks, blockSize>>>(lowerX, lowerY, resX, resY, stepX, stepY, d_data, maxIterations);

    hipMemcpy(data, d_data, size, hipMemcpyDeviceToHost);
    memcpy(img, data, size);
    hipFree(d_data);
    free(data);
}